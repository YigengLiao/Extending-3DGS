#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
    glm::vec3 pos = means[idx];
    glm::vec3 dir = pos - campos;
    dir = dir / glm::length(dir);

    glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

    glm::vec3 result = SH_C0 * sh[0];

    if (deg > 0)
    {
        float x = dir.x;
        float y = dir.y;
        float z = dir.z;
        result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

        if (deg > 1)
        {
            float xx = x * x, yy = y * y, zz = z * z;
            float xy = x * y, yz = y * z, xz = x * z;
            result = result +
                SH_C2[0] * xy * sh[4] +
                SH_C2[1] * yz * sh[5] +
                SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
                SH_C2[3] * xz * sh[7] +
                SH_C2[4] * (xx - yy) * sh[8];

            if (deg > 2)
            {
                result = result +
                    SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
                    SH_C3[1] * xy * z * sh[10] +
                    SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
                    SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
                    SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
                    SH_C3[5] * z * (xx - yy) * sh[14] +
                    SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
            }
        }
    }

    result += 0.5f;

    clamped[3 * idx + 0] = (result.x < 0);
    clamped[3 * idx + 1] = (result.y < 0);
    clamped[3 * idx + 2] = (result.z < 0);
    return glm::max(result, 0.0f);
}

__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float cov3D[6], const float* viewmatrix)
{

    float3 t = transformPoint4x3(mean, viewmatrix);

    const float limx = 1.3f * tan_fovx;
    const float limy = 1.3f * tan_fovy;
    const float txtz = t.x / t.z;
    const float tytz = t.y / t.z;
    t.x = min(limx, max(-limx, txtz)) * t.z;
    t.y = min(limy, max(-limy, tytz)) * t.z;

    glm::mat3 J = glm::mat3(
        focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
        0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
        0, 0, 0);

    glm::mat3 W = glm::mat3(
        viewmatrix[0], viewmatrix[4], viewmatrix[8],
        viewmatrix[1], viewmatrix[5], viewmatrix[9],
        viewmatrix[2], viewmatrix[6], viewmatrix[10]);

    glm::mat3 T = W * J;

    glm::mat3 Vrk = glm::mat3(
        cov3D[0], cov3D[1], cov3D[2],
        cov3D[1], cov3D[3], cov3D[4],
        cov3D[2], cov3D[4], cov3D[5]);

    glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

    return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float cov3D[6])
{

    glm::mat3 S = glm::mat3(1.0f);
    S[0][0] = mod * scale.x;
    S[1][1] = mod * scale.y;
    S[2][2] = mod * scale.z;

    glm::vec4 q = rot;
    float r = q.x;
    float x = q.y, y = q.z, z = q.w;

    glm::mat3 R = glm::mat3(
        1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
        2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
        2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
    );

    glm::mat3 M = S * R;

    glm::mat3 Sigma = glm::transpose(M) * M;

    cov3D[0] = Sigma[0][0];
    cov3D[1] = Sigma[0][1];
    cov3D[2] = Sigma[0][2];
    cov3D[3] = Sigma[1][1];
    cov3D[4] = Sigma[1][2];
    cov3D[5] = Sigma[2][2];
}

template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
    const float* orig_points,
    const glm::vec3* scales,
    const float scale_modifier,
    const glm::vec4* rotations,
    const float* opacities,
    const float* shs,
    bool* clamped,
    const float* cov3D_precomp,
    const float* colors_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const glm::vec3* cam_pos,
    const float* crop_box,
    const int W, int H, int tile_side,
    const float tan_fovx, float tan_fovy,
    const float focal_x, float focal_y,
    int* radii,
    float2* points_xy_image,
    float* depths,
    float* cov3Ds,
    float* rgb,
    float4* conic_opacity,
    float4* conic0_correction,
    int* point_id,
    float* v_dir,
    const dim3 grid,
    uint32_t* tiles_touched,
    bool prefiltered,
    int* P_counter)
{

    auto idx = cg::this_grid().thread_rank();
    if (idx >= P) return;
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

    float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
    float4 p_hom = transformPoint4x4(p_orig, projmatrix);

    float p_w = 1.0f / (p_hom.w + 1e-7f);
    float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

    float cov3D[6];
    if (cov3D_precomp != nullptr) {
        for (int i = 0; i < 6; ++i)
            cov3D[i] = cov3D_precomp[idx * 6 + i];
    } else {
        computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3D);
    }

    float3 cov0 = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);
    float3 cov = cov0;
    cov.x += 0.1f;
    cov.z += 0.1f;
    float det0 = (cov0.x * cov0.z - cov0.y * cov0.y);
    float det = (cov.x * cov.z - cov.y * cov.y);

    if (det0 == 0.0f || det == 0.0f) return;

    float det_inv = 1.f / det;
    float det_inv0 = 1.f / det0;
    float correction = sqrt(max(0.000025f, det0 * det_inv));
    float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };
    float3 conic0 = { cov0.z * det_inv0, -cov0.y * det_inv0, cov0.x * det_inv0 };

    float mid = 0.5f * (cov.x + cov.z);
    float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
    float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
    float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
    float2 point_image = { ndc2Pix(p_proj.x, W) - crop_box[0], ndc2Pix(p_proj.y, H) - crop_box[1] };

    uint2 rect_min, rect_max;
    getRect(point_image, my_radius, rect_min, rect_max, grid);
    if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0) return;

    unsigned mask    = __activemask();
    int leader       = __ffs(mask) - 1;
    int warp_count   = __popc(mask);

    int linear_tid = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);

    int lane = linear_tid & 31;

    int warp_base = (lane == leader) ? atomicAdd(P_counter, warp_count) : 0;
    warp_base = __shfl_sync(mask, warp_base, leader);

    unsigned lanemask_lt = (1u << lane) - 1u;
    int local_offset = __popc(mask & lanemask_lt);

    int out_idx = warp_base + local_offset;

    if (colors_precomp == nullptr)
    {
        glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
        rgb[out_idx * C + 0] = result.x;
        rgb[out_idx * C + 1] = result.y;
        rgb[out_idx * C + 2] = result.z;
    }

    cov3Ds[out_idx * 6 + 0] = cov3D[0];
    cov3Ds[out_idx * 6 + 1] = cov3D[1];
    cov3Ds[out_idx * 6 + 2] = cov3D[2];
    cov3Ds[out_idx * 6 + 3] = cov3D[3];
    cov3Ds[out_idx * 6 + 4] = cov3D[4];
    cov3Ds[out_idx * 6 + 5] = cov3D[5];

    depths[out_idx] = p_view.z;
    radii[out_idx] = my_radius;
    points_xy_image[out_idx] = point_image;
    conic_opacity[out_idx] = { conic.x, conic.y, conic.z, opacities[idx] };
    conic0_correction[out_idx] = { conic0.x, conic0.y, conic0.z, correction };
    tiles_touched[out_idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
    point_id[out_idx] = idx;

    v_dir[out_idx * 3 + 0] = cam_pos->x - p_orig.x;
    v_dir[out_idx * 3 + 1] = cam_pos->y - p_orig.y;
    v_dir[out_idx * 3 + 2] = cam_pos->z - p_orig.z;
}

template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
    const uint2* __restrict__ ranges,
    const uint32_t* __restrict__ point_list,
    int W, int H, int tile_side,
    const float2* __restrict__ points_xy_image,
    const float* __restrict__ features,
    const float4* __restrict__ conic_opacity,
    const float4* __restrict__ conic0_correction,
    float* __restrict__ final_W,
    const float* __restrict__ bg_color,
    float* __restrict__ out_color)
{
    auto block = cg::this_thread_block();
    uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
    uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
    uint2 pix_max = { min(pix_min.x + BLOCK_X, (uint32_t)W), min(pix_min.y + BLOCK_Y , (uint32_t)H) };
    uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
    uint32_t pix_id = W * pix.y + pix.x;
    float2 pixf = { (float)pix.x, (float)pix.y };
    bool inside = pix.x < (uint32_t)W && pix.y < (uint32_t)H;
    bool done = !inside;
    uint2 range;
    if (tile_side == 0){
        range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
    } else {
        range = ranges[block.group_index().z];
    }

    const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int toDo = range.y - range.x;

    __shared__ int   collected_id[BLOCK_SIZE];
    __shared__ float2 collected_xy[BLOCK_SIZE];
    __shared__ float4 collected_conic_opacity[BLOCK_SIZE];
    __shared__ float4 collected_conic0_correction[BLOCK_SIZE];

    float sum_w = 0.0f;
    float C[CHANNELS] = { 0 };

    for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
    {

        int progress = i * BLOCK_SIZE + block.thread_rank();
        if (range.x + progress < range.y)
        {
            int coll_id = point_list[range.x + progress];
            collected_id[block.thread_rank()] = coll_id;
            collected_xy[block.thread_rank()] = points_xy_image[coll_id];
            collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
            collected_conic0_correction[block.thread_rank()] = conic0_correction[coll_id];
        }
        block.sync();

        for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
        {

            float2 xy = collected_xy[j];
            float2 d = { xy.x - pixf.x, xy.y - pixf.y };
            float4 con_o = collected_conic_opacity[j];
            float correction = collected_conic0_correction[j].w;

            float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
            if (power > 0.0f) continue;
            float alpha = con_o.w * exp(power) * correction;
            //  float alpha = min(0.99f, con_o.w * exp(power) * correction);
            if (alpha < 1.0f / 255.0f) continue;
            for (int ch = 0; ch < CHANNELS; ch++)
                C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha;

            sum_w += alpha;
        }
    }
    const int pixel_offset = (tile_side == 0) ? 0 : (block.group_index().z * CHANNELS * H * W);
    const int pixel_offset_W = (tile_side == 0) ? 0 : (block.group_index().z * H * W);
    if (inside)
    {
        if (sum_w > 1.f) {
            const float invw = 1.f / sum_w;
            for (int ch = 0; ch < CHANNELS; ch++)
                out_color[pixel_offset + ch * H * W + pix_id] = C[ch] * invw;
        } else {
            const float fac = 1.f - sum_w;
            for (int ch = 0; ch < CHANNELS; ch++)
                out_color[pixel_offset + ch * H * W + pix_id] = C[ch] + fac * bg_color[ch];
        }
        final_W[pixel_offset_W + pix_id] = sum_w;
    }
}

void FORWARD::render(
    const dim3 grid, dim3 block,
    const uint2* ranges,
    const uint32_t* point_list,
    int W, int H, int tile_side,
    const float2* means2D,
    const float* colors,
    const float4* conic_opacity,
    const float4* conic0_correction,
    float* final_W,
    const float* bg_color,
    float* out_color)
{
    renderCUDA<NUM_CHANNELS> <<<grid, block>>>(
        ranges,
        point_list,
        W, H, tile_side,
        means2D,
        colors,
        conic_opacity,
        conic0_correction,
        final_W,
        bg_color,
        out_color);
}

void FORWARD::preprocess(int P, int D, int M,
    const float* means3D,
    const glm::vec3* scales,
    const float scale_modifier,
    const glm::vec4* rotations,
    const float* opacities,
    const float* shs,
    bool* clamped,
    const float* cov3D_precomp,
    const float* colors_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const glm::vec3* cam_pos,
    const float* crop_box,
    const int W, int H, int tile_side,
    const float focal_x, float focal_y,
    const float tan_fovx, float tan_fovy,
    int* radii,
    float2* means2D,
    float* depths,
    float* cov3Ds,
    float* rgb,
    float4* conic_opacity,
    float4* conic0_correction,
    int* point_id,
    float* v_dir,
    const dim3 grid,
    uint32_t* tiles_touched,
    bool prefiltered,
    int* P_counter)
{
    preprocessCUDA<NUM_CHANNELS> <<< (P + 255) / 256, 256 >>> (
        P, D, M,
        means3D,
        scales,
        scale_modifier,
        rotations,
        opacities,
        shs,
        clamped,
        cov3D_precomp,
        colors_precomp,
        viewmatrix,
        projmatrix,
        cam_pos,
        crop_box,
        W, H, tile_side,
        tan_fovx, tan_fovy,
        focal_x, focal_y,
        radii,
        means2D,
        depths,
        cov3Ds,
        rgb,
        conic_opacity,
        conic0_correction,
      	point_id,
      	v_dir,
        grid,
        tiles_touched,
        prefiltered,
      	P_counter
    );
}